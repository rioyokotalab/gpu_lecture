#include "hip/hip_runtime.h"
#include <cstdio>
#include <cstdlib>
#include <hip/hip_cooperative_groups.h>
using namespace cooperative_groups;

__global__ void scan(int *a, int *b, int N) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  grid_group grid = this_grid();
  for(int j=1; j<N; j<<=1) {
    b[i] = a[i];
    grid.sync();
    a[i] += b[i-j];
    grid.sync();
  }
}

int main() {
  const int N=8;
  int *a, *b;
  hipMallocManaged(&a, N*sizeof(int));
  hipMallocManaged(&b, N*sizeof(int));
  for(int i=0; i<N; i++) {
    a[i] = rand() & 3;
    printf("%*d ",2,a[i]);
  }
  printf("\n");
  void *args[] = {(void *)&a,  (void *)&b, (void *)&N};
  hipLaunchCooperativeKernel((void*)scan, 2, N/2, args);
  hipDeviceSynchronize();
  for(int i=0; i<N; i++)
    printf("%*d ",2,a[i]);
  printf("\n");
  hipFree(a);
  hipFree(b);
}
