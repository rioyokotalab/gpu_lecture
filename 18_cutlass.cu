
#include <hip/hip_runtime.h>
#include <iostream>
#include <typeinfo>
#include <random>
#include <stdint.h>
#include <hipblas.h>
#include <chrono>
using namespace std;

__global__ void kernel(int dim_m, int dim_n, int dim_k,
		       float *d_a, float *d_b, float *d_c) {
  const int ItemsPerVector = 4;
  const int VectorsPerThread = 2;
  const int ItemsPerThread = VectorsPerThread * ItemsPerVector; // 8

  const int ThreadsPerWarpY = 4;
  const int ThreadsPerWarpX = 8;
  const int ThreadsPerWarp = ThreadsPerWarpX * ThreadsPerWarpY; // 32
  const int WarpsPerBlockX = 1;
  const int ThreadsPerBlock = 64;

  const int ItemsPerWarpY = ThreadsPerWarpY * ItemsPerThread; // 32
  const int ItemsPerWarpX = ThreadsPerWarpX * ItemsPerThread; // 64
  const int ItemsPerBlockX = WarpsPerBlockX * ItemsPerWarpX; // 64

  const int Ktile = 8;
  const int VectorsPerMtile = ThreadsPerWarpX * VectorsPerThread; // 16 A #rows
  const int ThreadsPerKtile = ThreadsPerBlock / VectorsPerMtile; // 4 A #cols
  const int VectorsPerKtile = Ktile / ItemsPerVector; // 2 B #rows
  const int ThreadsPerNtile = ThreadsPerBlock / VectorsPerKtile; // 32 B #cols

  struct __align__(16) vec_t { float d[ItemsPerVector]; };
  vec_t *tile_a;
  vec_t *tile_b;
  vec_t __align__(16) thread_a[VectorsPerThread];
  vec_t __align__(16) thread_b[VectorsPerThread];
  __shared__ float __align__(16) block_a[Ktile][ItemsPerBlockX];
  __shared__ float __align__(16) block_b[Ktile][ItemsPerBlockX];
  float __align__(16) fragment_a[ItemsPerThread];
  float __align__(16) fragment_b[ItemsPerThread];
  float __align__(16) fragment_c[ItemsPerThread][ItemsPerThread];

  int offset_a_m = ItemsPerBlockX * blockIdx.x / ItemsPerVector;
  int offset_b_n = ItemsPerBlockX * blockIdx.y;
  int lda = dim_m / ItemsPerVector;
  int ldb = dim_k / ItemsPerVector;
  int a_m = threadIdx.x % VectorsPerMtile; // 16
  int a_k = threadIdx.x / VectorsPerMtile; // 4
  int b_k = threadIdx.x % VectorsPerKtile; // 2
  int b_n = threadIdx.x / VectorsPerKtile; // 32
  tile_a = reinterpret_cast<vec_t*>(&d_a[(a_k * lda + (a_m + offset_a_m)) * ItemsPerVector]);
  tile_b = reinterpret_cast<vec_t*>(&d_b[((b_n + offset_b_n) * ldb + b_k) * ItemsPerVector]);
  for (int m = 0; m < ItemsPerThread; ++m)
    for (int n = 0; n < ItemsPerThread; ++n)
      fragment_c[m][n] = 0;

  int warp_id = threadIdx.x / ThreadsPerWarp; // 2
  int warp_x = warp_id % WarpsPerBlockX; // 2
  int warp_y = warp_id / WarpsPerBlockX; // 1
  int lane_id = threadIdx.x % ThreadsPerWarp; // 32
  int lane_x = lane_id / ThreadsPerWarpY; // 8
  int lane_y = lane_id % ThreadsPerWarpY; // 4
  int offset_y = lane_y * ItemsPerVector + warp_y * ItemsPerWarpY; // 1*4*vec
  int offset_x = lane_x * ItemsPerVector + warp_x * ItemsPerWarpX; // 2*8*vec
  int stride_a = 0;
  int stride_b = 0;
  for (int kk = 0; kk < dim_k; kk += Ktile) {
    for (int i = 0; i < VectorsPerThread; ++i) {
      thread_a[i] = tile_a[stride_a + i * ThreadsPerKtile * lda]; // 2*a_k*vec
      thread_b[i] = tile_b[stride_b + i * ThreadsPerNtile * ldb]; // 2*b_n*vec
    }
    __syncthreads();
    for (int i = 0; i < VectorsPerThread; ++i) {
      for (int j = 0; j < ItemsPerVector; ++j) {                                         //    4       16
	block_a[a_k + i * ThreadsPerKtile][a_m * ItemsPerVector + j] = thread_a[i].d[j]; // (2*a_k) * (a_m*vec)
	block_b[b_k * ItemsPerVector + j][b_n + i * ThreadsPerNtile] = thread_b[i].d[j]; // (2*b_n) * (b_k*vec) ^T
      }                                                                                  //    32      2
    }
    __syncthreads();
    stride_a += lda * Ktile;
    stride_b += Ktile / ItemsPerVector;
#pragma unroll
    for (int k = 0; k < Ktile; k++) {
      for (int i = 0; i < VectorsPerThread; ++i) {
	for (int j = 0; j < ItemsPerVector; ++j) {
	  fragment_a[i * ItemsPerVector + j] = block_a[k][offset_y + (i * ThreadsPerWarpY * ItemsPerVector) + j]; // 4
	  fragment_b[i * ItemsPerVector + j] = block_b[k][offset_x + (i * ThreadsPerWarpX * ItemsPerVector) + j]; // 8
	}
      }
      for (int m = 0; m < ItemsPerThread; ++m) {
	for (int n = 0; n < ItemsPerThread; ++n) {
	  fragment_c[m][n] += fragment_a[m] * fragment_b[n];
	}
      }
    }
  }
  for (int ix = 0; ix < ItemsPerThread; ++ix) {
    for (int iy = 0; iy < ItemsPerThread; iy += ItemsPerVector) {
      int vx = ix / ItemsPerVector;
      int vy = iy / ItemsPerVector;
      int tx = offset_x + (vx * ThreadsPerWarpX * ItemsPerVector) + (ix % ItemsPerVector);
      int ty = offset_y + (vy * ThreadsPerWarpY * ItemsPerVector) + (iy % ItemsPerVector);
      int bx = ItemsPerBlockX * blockIdx.y + tx;
      int by = ItemsPerBlockX * blockIdx.x + ty;
      for (int i = 0; i < ItemsPerVector; ++i) {
	if (bx < dim_n && (by + i) < dim_m) {
	  d_c[bx * dim_m + by + i] = fragment_c[iy + i][ix];
	}
      }
    }
  }
}

int main(int argc, const char **argv) {
  int m = 10240;
  int k = 4096;
  int n = 8192;
  float alpha = 1.0;
  float beta = 0.0;
  int Nt = 10;
  float *A, *B, *C, *C2;
  hipMallocManaged(&A, m * k * sizeof(float));
  hipMallocManaged(&B, k * n * sizeof(float));
  hipMallocManaged(&C, m * n * sizeof(float));
  hipMallocManaged(&C2, m * n * sizeof(float));
  for (int i=0; i<m; i++)
    for (int j=0; j<k; j++)
      A[k*i+j] = drand48();
  for (int i=0; i<k; i++)
    for (int j=0; j<n; j++)
      B[n*i+j] = drand48();
  for (int i=0; i<n; i++)
    for (int j=0; j<m; j++)
      C[m*i+j] = C2[m*i+j] = 0;
  hipblasHandle_t cublas_handle;
  hipblasCreate(&cublas_handle);
  auto tic = chrono::steady_clock::now();
  for (int i = 0; i < Nt+2; i++) {
    if (i == 2) tic = chrono::steady_clock::now();
    hipblasSgemm(cublas_handle,
		HIPBLAS_OP_N,
		HIPBLAS_OP_N,
		m,
		n,
		k,
		&alpha,
		A,
		m,
		B,
		k,
		&beta,
		C,
		m);
    hipDeviceSynchronize();
  }
  auto toc = chrono::steady_clock::now();
  int64_t num_flops = (2 * int64_t(m) * int64_t(n) * int64_t(k)) + (2 * int64_t(m) * int64_t(n));
  double tcublas = chrono::duration<double>(toc - tic).count() / Nt;
  double cublas_flops = double(num_flops) / tcublas / 1.0e9;
  int tile = 64;
  dim3 block = dim3(tile);
  dim3 grid = dim3((m+tile-1)/tile, (n+tile-1)/tile);
  for (int i = 0; i < Nt+2; i++) {
    if (i == 2) tic = chrono::steady_clock::now();
    kernel<<< grid, block >>>(m,
			      n,
			      k,
			      A,
			      B,
			      C2);
    hipDeviceSynchronize();
  }
  toc = chrono::steady_clock::now();
  double tcutlass = chrono::duration<double>(toc - tic).count() / Nt;
  double cutlass_flops = double(num_flops) / tcutlass / 1.0e9;
  printf("CUBLAS: %.2f Gflops, CUTLASS: %.2f Gflops\n", cublas_flops, cutlass_flops);
  double err = 0;
  for (int i=0; i<n; i++) {
    for (int j=0; j<m; j++) {
      err += fabs(C[m*i+j] - C2[m*i+j]);
    }
  }
  printf("error: %lf\n", err/n/m);
  hipFree(A);
  hipFree(B);
  hipFree(C);
  hipFree(C2);
  hipblasDestroy(cublas_handle);
}
