
#include <hip/hip_runtime.h>
#include <cstdio>

__device__ __managed__ int sum;

__device__ int warpSum(int a) {
  for (int offset=16; offset>0; offset >>= 1)
    a += __shfl_down_sync(0xffffffff, a, offset);
  return a;
}

__global__ void reduction(int &sum, int *a) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  int b = warpSum(a[i]);
  if ((threadIdx.x & 31) == 0)
    atomicAdd(&sum, b);
}

int main(void) {
  const int N = 128;
  const int M = 64;
  int *a;
  hipMallocManaged(&a, N*sizeof(int));
  for (int i=0; i<N; i++) a[i] = 1;
  reduction<<<N/M,M>>>(sum, a);
  hipDeviceSynchronize();
  printf("%d\n",sum);
  hipFree(a);
}
