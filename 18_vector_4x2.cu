
#include <hip/hip_runtime.h>
#include <iostream>
#include <typeinfo>
#include <random>
#include <stdint.h>
#include <hipblas.h>
#include <chrono>
using namespace std;

__global__ void kernel(int dim_m, int dim_n, int dim_k,
		       float *d_a, float *d_b, float *d_c) {
  int offset_a_m = 64 * blockIdx.x;
  int offset_b_n = 64 * blockIdx.y;
  int a_m = threadIdx.x % 16 * 4;
  int a_k = threadIdx.x / 16;
  int b_k = threadIdx.x % 2 * 4;
  int b_n = threadIdx.x / 2;

  struct __align__(16) vec_t { float d[4]; };
  __shared__ float __align__(16) block_a[8][64];
  __shared__ float __align__(16) block_b[8][64];
  float block_c[8][8];
  vec_t thread_a[2];
  vec_t thread_b[2];

  vec_t *tile_a = reinterpret_cast<vec_t*>(&d_a[a_k * dim_m + (a_m + offset_a_m)]);
  vec_t *tile_b = reinterpret_cast<vec_t*>(&d_b[(b_n + offset_b_n) * dim_k + b_k]);
  for (int m = 0; m < 8; ++m)
    for (int n = 0; n < 8; ++n)
      block_c[m][n] = 0;

  int warp_id = threadIdx.x / 32;
  int lane_id = threadIdx.x % 32;
  int lane_n = lane_id / 4;
  int lane_m = lane_id % 4;
  int offset_n = lane_n * 4;
  int offset_m = warp_id * 32 + lane_m * 4;
  int offset_a_k = 0;
  int offset_b_k = 0;
  for (int k = 0; k < dim_k; k += 8) {
    for (int i = 0; i < 2; ++i) {
      thread_a[i] = tile_a[offset_a_k + i * dim_m];
      thread_b[i] = tile_b[offset_b_k + i * 8 * dim_k];
    }
    __syncthreads();
    for (int i = 0; i < 2; ++i) {
      for (int j = 0; j < 4; ++j) {
	block_a[a_k + i * 4][a_m + j] = thread_a[i].d[j];
	block_b[b_k+ j][b_n + i * 32] = thread_b[i].d[j];
      }
    }
    __syncthreads();
    offset_a_k += dim_m * 2;
    offset_b_k += 2;
#pragma unroll
    for (int j = 0; j < 8; ++j) {
      for (int m = 0; m < 8; ++m) {
	for (int n = 0; n < 8; ++n) {
	  block_c[m][n] += block_a[j][offset_m + m / 4 * 16 + m % 4] * block_b[j][offset_n + n / 4 * 32 + n % 4];
	}
      }
    }
  }
  for (int m = 0; m < 8; ++m) {
    for (int n = 0; n < 8; ++n) {
      int c_n = offset_b_n + offset_n + n / 4 * 32 + n % 4;
      int c_m = offset_a_m + offset_m + m / 4 * 16 + m % 4;
      if (c_n < dim_n && c_m < dim_m) {
	d_c[c_n * dim_m + c_m] = block_c[m][n];
      }
    }
  }
}

int main(int argc, const char **argv) {
  int m = 10240;
  int k = 4096;
  int n = 8192;
  float alpha = 1.0;
  float beta = 0.0;
  int Nt = 10;
  float *A, *B, *C, *C2;
  hipMallocManaged(&A, m * k * sizeof(float));
  hipMallocManaged(&B, k * n * sizeof(float));
  hipMallocManaged(&C, m * n * sizeof(float));
  hipMallocManaged(&C2, m * n * sizeof(float));
  for (int i=0; i<m; i++)
    for (int j=0; j<k; j++)
      A[k*i+j] = drand48();
  for (int i=0; i<k; i++)
    for (int j=0; j<n; j++)
      B[n*i+j] = drand48();
  for (int i=0; i<n; i++)
    for (int j=0; j<m; j++)
      C[m*i+j] = C2[m*i+j] = 0;
  hipblasHandle_t cublas_handle;
  hipblasCreate(&cublas_handle);
  auto tic = chrono::steady_clock::now();
  for (int i = 0; i < Nt+2; i++) {
    if (i == 2) tic = chrono::steady_clock::now();
    hipblasSgemm(cublas_handle,
		HIPBLAS_OP_N,
		HIPBLAS_OP_N,
		m,
		n,
		k,
		&alpha,
		A,
		m,
		B,
		k,
		&beta,
		C,
		m);
    hipDeviceSynchronize();
  }
  auto toc = chrono::steady_clock::now();
  int64_t num_flops = (2 * int64_t(m) * int64_t(n) * int64_t(k)) + (2 * int64_t(m) * int64_t(n));
  double tcublas = chrono::duration<double>(toc - tic).count() / Nt;
  double cublas_flops = double(num_flops) / tcublas / 1.0e9;
  int tile = 64;
  dim3 block = dim3(tile);
  dim3 grid = dim3((m+tile-1)/tile, (n+tile-1)/tile);
  for (int i = 0; i < Nt+2; i++) {
    if (i == 2) tic = chrono::steady_clock::now();
    kernel<<< grid, block >>>(m,
			      n,
			      k,
			      A,
			      B,
			      C2);
    hipDeviceSynchronize();
  }
  toc = chrono::steady_clock::now();
  double tcutlass = chrono::duration<double>(toc - tic).count() / Nt;
  double cutlass_flops = double(num_flops) / tcutlass / 1.0e9;
  printf("CUBLAS: %.2f Gflops, CUTLASS: %.2f Gflops\n", cublas_flops, cutlass_flops);
  double err = 0;
  for (int i=0; i<n; i++) {
    for (int j=0; j<m; j++) {
      err += fabs(C[m*i+j] - C2[m*i+j]);
    }
  }
  printf("error: %lf\n", err/n/m);
  hipFree(A);
  hipFree(B);
  hipFree(C);
  hipFree(C2);
  hipblasDestroy(cublas_handle);
}
