
#include <hip/hip_runtime.h>
#include <cstdio>

__device__ __managed__ int sum;

__global__ void reduction(int &sum, int *a) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  extern __shared__ int b[];
  __syncthreads();
  b[threadIdx.x] = a[i];
  __syncthreads();
  int c = 0;
  for (int j=0; j<blockDim.x; j++)
    c += b[j];
  if (threadIdx.x == 0)
    atomicAdd(&sum, c);
}

int main(void) {
  const int N = 128;
  const int M = 64;
  int *a;
  hipMallocManaged(&a, N*sizeof(int));
  for (int i=0; i<N; i++) a[i] = 1;
  reduction<<<N/M,M,M*sizeof(int)>>>(sum, a);
  hipDeviceSynchronize();
  printf("%d\n",sum);
  hipFree(a);
}
