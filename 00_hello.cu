
#include <hip/hip_runtime.h>
#include <cstdio>

__global__ void print(void) {
  printf("Hello GPU\n");
}

int main() {
  printf("Hello CPU\n");
  print<<<1,1>>>();
  hipDeviceSynchronize();
}
