#include "hip/hip_runtime.h"
#include <mpi.h>
#include <cstdio>

__global__ void GPU_Kernel() {
  printf(" GPU block  : %d / %d  GPU thread : %d / %d\n",
         blockIdx.x, gridDim.x, threadIdx.x, blockDim.x);
}

int main(int argc, char **argv) {
  char hostname[256];
  int mpisize, mpirank, gpusize, gpurank, len;
  MPI_Init(&argc, &argv);
  MPI_Comm_size(MPI_COMM_WORLD, &mpisize);
  MPI_Comm_rank(MPI_COMM_WORLD, &mpirank);
  MPI_Get_processor_name(hostname, &len);
  hipGetDeviceCount(&gpusize);
  hipSetDevice(mpirank % gpusize);
  hipGetDevice(&gpurank);
  for (int irank=0; irank<mpisize; irank++) {
    MPI_Barrier(MPI_COMM_WORLD);
    if (mpirank == irank) {
      printf("Hostname    : %s\n", hostname);
      printf("MPI rank    : %d / %d  GPU device : %d / %d\n",
             mpirank, mpisize, gpurank, gpusize);
      GPU_Kernel<<<2,2>>>();
      hipDeviceSynchronize();
    }
  }
  MPI_Finalize();
}
