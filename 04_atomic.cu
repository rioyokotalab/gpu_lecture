
#include <hip/hip_runtime.h>
#include <cstdio>

__device__ __managed__ int sum;

__global__ void reduction(int &sum) {
  //sum += 1;
  atomicAdd(&sum, 1);
}

int main(void) {
  const int N = 128;
  const int M = 64;
  reduction<<<N/M,M>>>(sum);
  hipDeviceSynchronize();
  printf("%d\n",sum);
}
