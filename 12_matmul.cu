
#include <hip/hip_runtime.h>
#include <iostream>
#include <typeinfo>
#include <random>
#include <stdint.h>
#include <hipblas.h>
#include <chrono>
using namespace std;

__global__ void kernel(int dim_m, int dim_n, int dim_k,
		       float *d_a, float *d_b, float *d_c) {
  int offset_a_m = 64 * blockIdx.x;
  int offset_b_n = 64 * blockIdx.y;
  int m = threadIdx.x;

  float block_c[64];

  for (int n = 0; n < 64; ++n)
    block_c[n] = 0;

  for (int k = 0; k < dim_k; k += 8) {
    int offset_a_k = k, offset_b_k = k;
    for (int j = 0; j < 8; ++j) {
      float block_a = d_a[(offset_a_k + j) * dim_m + offset_a_m + m];
      for (int n = 0; n < 64; ++n) {
        float block_b = d_b[(offset_b_n + n) * dim_k + offset_b_k + j];
	block_c[n] += block_a * block_b;
      }
    }
  }
  for (int n = 0; n < 64; ++n) {
    int c_n = offset_b_n + n;
    int c_m = offset_a_m + m;
    if (c_n < dim_n && c_m < dim_m) {
      d_c[c_n * dim_m + c_m] = block_c[n];
    }
  }
}

int main(int argc, const char **argv) {
  int m = 10240;
  int k = 4096;
  int n = 8192;
  float alpha = 1.0;
  float beta = 0.0;
  int Nt = 10;
  float *A, *B, *C, *C2;
  hipMallocManaged(&A, m * k * sizeof(float));
  hipMallocManaged(&B, k * n * sizeof(float));
  hipMallocManaged(&C, m * n * sizeof(float));
  hipMallocManaged(&C2, m * n * sizeof(float));
  for (int i=0; i<m; i++)
    for (int j=0; j<k; j++)
      A[k*i+j] = drand48();
  for (int i=0; i<k; i++)
    for (int j=0; j<n; j++)
      B[n*i+j] = drand48();
  for (int i=0; i<n; i++)
    for (int j=0; j<m; j++)
      C[m*i+j] = C2[m*i+j] = 0;
  hipblasHandle_t cublas_handle;
  hipblasCreate(&cublas_handle);
  auto tic = chrono::steady_clock::now();
  for (int i = 0; i < Nt+2; i++) {
    if (i == 2) tic = chrono::steady_clock::now();
    hipblasSgemm(cublas_handle,
		HIPBLAS_OP_N,
		HIPBLAS_OP_N,
		m,
		n,
		k,
		&alpha,
		A,
		m,
		B,
		k,
		&beta,
		C,
		m);
    hipDeviceSynchronize();
  }
  auto toc = chrono::steady_clock::now();
  int64_t num_flops = (2 * int64_t(m) * int64_t(n) * int64_t(k)) + (2 * int64_t(m) * int64_t(n));
  double tcublas = chrono::duration<double>(toc - tic).count() / Nt;
  double cublas_flops = double(num_flops) / tcublas / 1.0e9;
  int tile = 64;
  dim3 block = dim3(tile);
  dim3 grid = dim3((m+tile-1)/tile, (n+tile-1)/tile);
  for (int i = 0; i < Nt+2; i++) {
    if (i == 2) tic = chrono::steady_clock::now();
    kernel<<< grid, block >>>(m,
			      n,
			      k,
			      A,
			      B,
			      C2);
    hipDeviceSynchronize();
  }
  toc = chrono::steady_clock::now();
  double tcutlass = chrono::duration<double>(toc - tic).count() / Nt;
  double cutlass_flops = double(num_flops) / tcutlass / 1.0e9;
  printf("CUBLAS: %.2f Gflops, CUTLASS: %.2f Gflops\n", cublas_flops, cutlass_flops);
  double err = 0;
  for (int i=0; i<n; i++) {
    for (int j=0; j<m; j++) {
      err += fabs(C[m*i+j] - C2[m*i+j]);
    }
  }
  printf("error: %lf\n", err/n/m);
  hipFree(A);
  hipFree(B);
  hipFree(C);
  hipFree(C2);
  hipblasDestroy(cublas_handle);
}
