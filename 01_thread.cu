
#include <hip/hip_runtime.h>
#include <cstdio>

__global__ void thread(float *a) {
  a[threadIdx.x] = threadIdx.x;
}

int main(void) {
  const int N = 4;
  float *a;
  hipMallocManaged(&a, N*sizeof(float));
  thread<<<1,N>>>(a);
  hipDeviceSynchronize();
  for (int i=0; i<N; i++)
    printf("%d %g\n",i,a[i]);
  hipFree(a);
}