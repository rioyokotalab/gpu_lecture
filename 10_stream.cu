
#include <hip/hip_runtime.h>
#include <cstdio>

__global__ void block(float *a, int N) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if(i>=N) return;
  a[i] = i;
}

int main(void) {
  const int N = 2000;
  const int M = 1024;
  const int num_streams = 8;
  float *a;
  hipStream_t streams[num_streams];
  hipMallocManaged(&a, N*sizeof(float));
  for (int i=0; i<num_streams; i++) {
    hipStreamCreate(&streams[i]);
    block<<<(N+M-1)/M,M,0,streams[i]>>>(a,N);
  }
  hipDeviceSynchronize();
  for (int i=0; i<N; i++)
    printf("%d %g\n",i,a[i]);
  hipFree(a);
}
