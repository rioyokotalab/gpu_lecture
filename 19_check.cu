
#include <hip/hip_runtime.h>
#include <cstdio>

void cudaCheckError() {
  hipError_t err=hipGetLastError();
  if(err!=hipSuccess) {
    fprintf(stderr,"CUDA failure %s:%d: '%s'\n",__FILE__,__LINE__,hipGetErrorString(err));
    exit(-1);
  }
}

__global__ void thread(float *a) {
  a[threadIdx.x] = threadIdx.x;
}

int main(void) {
  const int N = 2048;
  float *a;
  hipMallocManaged(&a, N*sizeof(float));
  thread<<<1,N>>>(a);
  cudaCheckError();
  for (int i=0; i<N; i++)
    printf("%d %g\n",i,a[i]);
  hipFree(a);
}
