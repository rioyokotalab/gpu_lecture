
#include <hip/hip_runtime.h>
#include <cstdio>

__global__ void oddeven(float *a) {
  int i = threadIdx.x;
  if (i & 1)
    a[i] = -i;
  else
    a[i] = i;
}

int main(void) {
  const int N = 32;
  float *a;
  hipMallocManaged(&a, N*sizeof(float));
  oddeven<<<1,N>>>(a);
  hipDeviceSynchronize();
  for (int i=0; i<N; i++)
    printf("%d %g\n",i,a[i]);
  hipFree(a);
}