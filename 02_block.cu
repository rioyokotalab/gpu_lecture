
#include <hip/hip_runtime.h>
#include <cstdio>

__global__ void block(float *a, int N) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if(i>=N) return;
  a[i] = i;
}

int main(void) {
  const int N = 2000;
  const int M = 1024;
  float *a;
  hipMallocManaged(&a, N*sizeof(float));
  block<<<(N+M-1)/M,M>>>(a,N);
  hipDeviceSynchronize();
  for (int i=0; i<N; i++)
    printf("%d %g\n",i,a[i]);
  hipFree(a);
}