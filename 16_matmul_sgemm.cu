
#include <hip/hip_runtime.h>
#include <cmath>
#include <hipblas.h>
#include <cstdlib>
#include <cstdio>
#include <chrono>
using namespace std;

#define M 1024

int main(int argc, char **argv) {
  int N = 2048;
  int size = N * N * sizeof(float);
  float *A, *B, *C;
  hipMallocManaged(&A, size);
  hipMallocManaged(&B, size);
  hipMallocManaged(&C, size);
  for (int i=0; i<N; i++) {
    for (int j=0; j<N; j++) {
      A[N*i+j] = drand48();
      B[N*i+j] = drand48();
      C[N*i+j] = 0;
    }
  }
  float alpha = 1.0;
  float beta = 0.0;
  hipblasHandle_t handle;
  hipblasStatus_t stat = hipblasCreate(&handle);
  stat = hipblasSetMatrix(N, N, sizeof(*A), A, N, A, N);
  stat = hipblasSetMatrix(N, N, sizeof(*B), B, N, B, N);
  stat = hipblasSetMatrix(N, N, sizeof(*C), C, N, C, N);
  auto tic = chrono::steady_clock::now();
  stat = hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, N, N,
                     &alpha, B, N, A, N, &beta, C, N);
  hipDeviceSynchronize();
  auto toc = chrono::steady_clock::now();
  stat = hipblasGetMatrix(N, N, sizeof(*C), C, N, C, N);
  double time = chrono::duration<double>(toc - tic).count();
  printf("N=%d: %lf s (%lf GFlops)\n",N,time,2.*N*N*N/time/1e9);
#pragma omp parallel for
  for (int i=0; i<N; i++)
    for (int k=0; k<N; k++)
      for (int j=0; j<N; j++)
        C[N*i+j] -= A[N*i+k] * B[N*k+j];
  double err = 0;
  for (int i=0; i<N; i++)
    for (int j=0; j<N; j++)
      err += fabs(C[N*i+j]);
  printf("error: %lf\n",err/N/N);
  hipFree(A);
  hipFree(B);
  hipFree(C);
  hipblasDestroy(handle);
}
