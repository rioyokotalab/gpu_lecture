
#include <hip/hip_runtime.h>
#include <cstdio>
#include <cstdlib>

__global__ void scan(int *a, int *b, int N) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  for(int j=1; j<N; j<<=1) {
    b[i] = a[i];
    __syncthreads();
    if(i>=j) a[i] += b[i-j];
    __syncthreads();
  }
}

int main() {
  const int N=8;
  int *a, *b;
  hipMallocManaged(&a, N*sizeof(int));
  hipMallocManaged(&b, N*sizeof(int));
  for(int i=0; i<N; i++) {
    a[i] = rand() & 3;
    printf("%*d ",2,a[i]);
  }
  printf("\n");
  scan<<<1,N>>>(a, b, N);
  hipDeviceSynchronize();
  for(int i=0; i<N; i++)
    printf("%*d ",2,a[i]);
  printf("\n");
  hipFree(a);
  hipFree(b);
}
